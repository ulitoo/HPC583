#include <cblas.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <iostream>
#include <thread>
#include <lapacke.h>
#include <fstream>
#include <random>
#include <hip/hip_runtime.h>
#include "magma_lapack.h"
#include "magma_v2.h"

int main(int argc, char *argv[])
{
    if (argc != 2)
    {
        std::cerr << "Usage: " << argv[0] << " n (Dimension of Matrix)" << std::endl;
        return 1;
    }
    magma_init(); // initialize Magma
    real_Double_t gpu_time;
    magma_int_t *piv, info;
    magma_int_t m; // a - mxm matrix
    magma_int_t n; // c - mxn matrix
    m = std::atoi(argv[1]);
    n = std::atoi(argv[1]);
    magma_int_t mm = m * m;              // size of a
    magma_int_t mn = m * n;              // size of c
    float *a;                            // a- mxm matrix on the host
    float *b;                            // b- mxn matrix on the host
    float *c;                            // c- mxn matrix on the host
    float *d_a;                          // d_a - mxm matrix a on the device
    float *d_c;                          // d_c - mxn matrix c on the device
    magma_int_t ione = 1;                // random uniform distr . in (0 ,1)
    magma_int_t ISEED[4] = {0, 0, 0, 1}; // seed
    magma_int_t err;
    const float alpha = 1.0; // alpha =1
    const float beta = 0.0;  // beta =0
    magma_queue_t queue = NULL;
    magma_int_t dev = 0;
    magma_queue_create(dev, &queue);

    // allocate matrices on the host
    err = magma_smalloc_pinned(&a, mm); // host memory for a
    err = magma_smalloc_pinned(&b, mn); // host memory for b
    err = magma_smalloc_pinned(&c, mn); // host memory for c
    err = magma_smalloc(&d_a, mm);      // device memory for a
    err = magma_smalloc(&d_c, mn);      // device memory for c
    piv = (magma_int_t *)malloc(m * sizeof(magma_int_t));

    // generate random matrices a, b;
    // lapackf77_slarnv(&ione, ISEED, &mm, a); // randomize a
    LAPACK_slarnv(&ione, ISEED, &mm, a); // randomize a
    // lapackf77_slarnv(&ione, ISEED, &mn, b); // randomize b
    LAPACK_slarnv(&ione, ISEED, &mn, b); // randomize b
    printf(" upper left corner of the expected solution :\n");
    magma_sprint(4, 4, b, m);

    // right hand side c=a*b
    // blasf77_sgemm("N", "N", &m, &n, &n, &alpha, a, &m, b, &m, &beta, c, &m);
    cblas_sgemm(CblasColMajor, CblasNoTrans, CblasNoTrans, m, n, n, alpha, a, m, b, m, beta, c, m);
    magma_ssetmatrix(m, m, a, m, d_a, m, queue); // copy a -> d_a
    magma_ssetmatrix(m, n, c, m, d_c, m, queue); // copy c -> d_c

    // MAGMA
    // solve the linear system d_a *x=d_c , d_a -mxm matrix ,
    // d_c -mxn matrix , d_c is overwritten by the solution ;
    // LU decomposition with partial pivoting and row
    // interchanges is used , row i is interchanged with row piv (i)

    gpu_time = magma_sync_wtime(NULL);
    // magma_sgesv(m, n, a, m, piv, c, m, &info);
    magma_sgesv_gpu(m, n, d_a, m, piv, d_c, m, &info);
    gpu_time = magma_sync_wtime(NULL) - gpu_time;
    printf(" magma_sgesv_gpu time : %7.5f sec .\n", gpu_time); // time
    magma_sgetmatrix(m, n, d_c, m, c, m, queue);
    printf(" upper left corner of the solution :\n");
    magma_sprint(4, 4, c, m); // part of the solution

    printf("\n Error : %i.\n", err); // time

    magma_free_pinned(a);       // free host memory
    magma_free_pinned(b);       // free host memory
    magma_free_pinned(c);       // free host memory
    free(piv);                  // free host memory
    magma_free(d_a);            // free device memory
    magma_free(d_c);            // free device memory
    magma_queue_destroy(queue); // destroy queue
    magma_finalize();           // finalize Magma
    return 0;
}
