
#include <hip/hip_runtime.h>
#include <iostream>

// CUDA kernel to print message from each thread
__global__ void helloCUDA()
{
    int threadId = threadIdx.x;
    int totalThreads = blockDim.x * gridDim.x;

    printf("Hello, World! from thread %d out of %d. GRID DIM %d and BLOCK DIM %d\n", threadId, totalThreads, blockDim.x, gridDim.x);
    //printf("GRID DIM %d and BLOCK DIM %d\n", blockDim.x, gridDim.x);
}

int main()
{
    // Specify the number of blocks and threads per block
    int numBlocks = 2; // You can change this to the desired number of blocks (Block Dimension)
    int threadsPerBlock = 5; // You can change this to the desired number of threads per block (Grid Dimension)

    // Launch the helloCUDA kernel with the specified number of blocks and threads per block
    helloCUDA<<<numBlocks, threadsPerBlock>>>();

    // Synchronize to make sure the kernel is done before exiting
    hipDeviceSynchronize();

    // Check for errors during kernel launch
    hipError_t hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(hipError_t) << std::endl;
        return -1;
    }

    std::cout << "CUDA Kernel executed successfully!" << std::endl;

    return 0;
}
