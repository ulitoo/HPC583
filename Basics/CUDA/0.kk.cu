#include <iostream>
#include "hip/hip_runtime.h"
#include "hipsolver.h"

void solveLinearSystem(const double *A, const double *B, double *X, int N, int numSystems) {
    hipsolverHandle_t cusolverH;
    hipsolverDnCreate(&cusolverH);

    int *devInfo;
    hipMalloc((void**)&devInfo, numSystems * sizeof(int));

    // Prepare pivoting array
    int *d_pivotArray;
    hipMalloc((void**)&d_pivotArray, N * numSystems * sizeof(int));

    // Prepare workspace
    size_t workspaceSize;
    cusolverDnDgetrsBatched_bufferSize(cusolverH, HIPBLAS_OP_N, N, 1, &workspaceSize, numSystems);
    double *d_workspace;
    hipMalloc((void**)&d_workspace, workspaceSize);

    // Solve the systems
    cusolverDnDgetrsBatched(cusolverH, HIPBLAS_OP_N, N, 1, A, N, d_pivotArray, B, N, X, N, devInfo, numSystems);

    hipFree(devInfo);
    hipFree(d_pivotArray);
    hipFree(d_workspace);
    hipsolverDnDestroy(cusolverH);
}

int main() {
    const int N = 3; // Size of the matrix
    const int numSystems = 3; // Number of linear systems

    // Define the coefficient matrix A and the matrices B and X
    double A[numSystems][N][N] = {
        {{6, -2, 2}, {4, 2, 5}, {2, 8, 7}},
        {{1, 2, 3}, {4, 5, 6}, {7, 8, 9}},
        {{3, 2, 1}, {6, 5, 4}, {9, 8, 7}}
    };
    double B[numSystems][N][N] = {
        {{16, 35, 64}, {16, 35, 64}, {16, 35, 64}},
        {{10, 20, 30}, {10, 20, 30}, {10, 20, 30}},
        {{1, 0, 0}, {0, 1, 0}, {0, 0, 1}}
    };
    double X[numSystems][N][N]; // Solution matrices

    // Copy matrices A and B to device memory
    double *d_A, *d_B, *d_X;
    hipMalloc((void**)&d_A, N * N * numSystems * sizeof(double));
    hipMalloc((void**)&d_B, N * N * numSystems * sizeof(double));
    hipMemcpy(d_A, A, N * N * numSystems * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * N * numSystems * sizeof(double), hipMemcpyHostToDevice);

    // Solve the linear systems
    solveLinearSystem(d_A, d_B, reinterpret_cast<double*>(X), N, numSystems);

    // Copy the solutions back to the host
    hipMemcpy(X, d_X, N * N * numSystems * sizeof(double), hipMemcpyDeviceToHost);

    // Print the solution matrices
    std::cout << "Solution matrices X:\n";
    for (int i = 0; i < numSystems; ++i) {
        std::cout << "System " << i + 1 << ":\n";
        for (int j = 0; j < N; ++j) {
            for (int k = 0; k < N; ++k) {
                std::cout << X[i][j][k] << " ";
            }
            std::cout << "\n";
        }
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_X);

    return 0;
}
