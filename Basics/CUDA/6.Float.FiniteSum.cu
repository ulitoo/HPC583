#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thread>
#include <random>
#include "0.CUDA_functions.h"
#define MI_BLOCKSIZE 256
using namespace std;

__global__ void reduce0(float *g_idata, float *g_odata)
{
    extern __shared__ float sdata[];
    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    sdata[tid] = g_idata[i];
    //printf("%f ",sdata[tid]);
    //printf("%d ",blockDim.x);
    __syncthreads();
    printf("%f ",sdata[tid]);
    //printf("%f ",g_idata[i]);
    // do reduction in shared mem
    for (unsigned int s = 1; s < blockDim.x; s *= 2)
    {
        //printf("%d - ",s);
        if (tid % (2 * s) == 0)
        {
            sdata[tid] += sdata[tid + s];
            //printf("%f",sdata[tid]);
            //printf("%d - ",s);
        }
        __syncthreads();
    }
    // write result for this block to global mem
    
    if (tid == 0)
        {g_odata[blockIdx.x] = sdata[0];
        //printf("%f\n",sdata[0]);
        }
}

int main(int argc, char *argv[])
{
    if (argc != 3)
    {
        std::cerr << "Usage of Finite SUM " << argv[0] << " 1/r (factor) n (Dimension)" << std::endl;
        return 1;
    }
    const float r = 1.0 / (float)std::atoi(argv[1]);
    const int size = std::atoi(argv[2]);

    auto start = std::chrono::high_resolution_clock::now();
    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start);
    long double elapsed_time, elapsed_time2;

    // Create a random number generator =>  Get a Seed from random device
    std::mt19937_64 rng(13);
    std::uniform_real_distribution<double> dist(0.0, 1.0); 

    float *a = new float[size];

    init_sum2 (a,r,size);
    float sum_exact = finitesum_exact(r,size); 

    float *dev_a;
    float *dev_c;
    hipMalloc((void **)&dev_a, size * sizeof(float));

    //start = std::chrono::high_resolution_clock::now();
    hipMemcpy(dev_a, a, size * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = MI_BLOCKSIZE;
    int gridSize = (size + blockSize - 1) / blockSize;
    hipMalloc((void **)&dev_c, gridSize * sizeof(float));
    float *c_gpu = new float[gridSize];

    start = std::chrono::high_resolution_clock::now();
    
    ///////////////////////////////////////////////////////////////////////////////
    //finitesum_GPU<<<gridSize, blockSize>>>(dev_a, dev_c, size);
    reduce0<<<gridSize, blockSize>>>(dev_a, dev_c);
    //printf("%f",dev_c[0]);
    ///////////////////////////////////////////////////////////////////////////////

    //stop = std::chrono::high_resolution_clock::now();
    hipMemcpy(c_gpu, dev_c, gridSize * sizeof(float), hipMemcpyDeviceToHost);
    
    for (unsigned int k = 0; k < 1; k++)
    {
        std::cout<< c_gpu[k] << " *** ";
        //std::cout<< a[k] << " ";
    }
    
    stop = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start);
    elapsed_time = duration.count() * 1.e-9;
    std::cout << "GPU time:" << elapsed_time << std::endl;

    hipFree(dev_a);
    hipFree(dev_c);

    start = std::chrono::high_resolution_clock::now();
    float sum_CPU = finitesum_CPU(a,size);
    stop = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start);
    elapsed_time2 = duration.count() * 1.e-9;
    std::cout << "CPU time:" << elapsed_time2 << std::endl;
    std::cout << "GPU speedup is x" << elapsed_time2 / elapsed_time << std::endl;
    std::cout << "GRIDSIZE:" << gridSize << std::endl;

    std::cout << "Exact results:" << sum_exact << " / GPU results: " << c_gpu[0] << " / CPU results: " << sum_CPU << std::endl;
    std::cout << "diff CPU:" << sum_exact-sum_CPU << " / diff GPU: " << sum_exact-c_gpu[0] << std::endl;

    delete[] a;
    delete[] c_gpu;

    return 0;
}
