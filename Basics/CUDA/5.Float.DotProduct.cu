#include <iostream>
#include <hip/hip_runtime.h>

#include <thread>
#include <random>
#define MI_BLOCKSIZE 256
using namespace std;

__global__ void dotproductGPU(float *a, float *b, float *c, int size)
{
    __shared__ float temp[MI_BLOCKSIZE];
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int localIndex = threadIdx.x;

    if (localIndex < size)
    {
        temp[localIndex] = a[tid] * b[tid];
    }
    __syncthreads();

    // Store the result in c Collecting from all Blocks with Atomic Add 
    if (localIndex == 0)
    {
        float sum = 0;
        for (int i = 0; i < MI_BLOCKSIZE; i++)
        {
            sum += temp[i]; 
        }
        atomicAdd(c,sum);
    }
}

float dotproductCPU(float *a, float *b, int size)
{
    float dot = 0.0;
    for (int i = 0; i < size; ++i)
    {
        dot += a[i] * b[i];
    }
    return dot;
}

int main(int argc, char *argv[])
{
    if (argc != 2)
    {
        std::cerr << "Usage of Dot Product: " << argv[0] << " n (Dimension of vector)" << std::endl;
        return 1;
    }
    const int size = std::atoi(argv[1]);

    auto start = std::chrono::high_resolution_clock::now();
    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start);
    long double elapsed_time, elapsed_time2;

    // Create a random number generator =>  Get a Seed from random device
    std::mt19937_64 rng(13);
    std::uniform_real_distribution<double> dist(0.0, 1.0); 

    float dotCPU = 0.0;
    float *a = new float[size];
    float *b = new float[size];
    float *c_gpu = new float[1];


    for (int i = 0; i < size; ++i)
    {
        a[i] = dist(rng) - 0.5;
        b[i] = dist(rng) - 0.5;
    }

    float *dev_a;
    float *dev_b;
    float *dev_c;
    hipMalloc((void **)&dev_a, size * sizeof(float));
    hipMalloc((void **)&dev_b, size * sizeof(float));
    hipMalloc((void **)&dev_c, sizeof(float));

    //start = std::chrono::high_resolution_clock::now();
    hipMemcpy(dev_a, a, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = MI_BLOCKSIZE;
    int gridSize = (size + blockSize - 1) / blockSize;
    start = std::chrono::high_resolution_clock::now();
    dotproductGPU<<<gridSize, blockSize>>>(dev_a, dev_b, dev_c, size);
    //stop = std::chrono::high_resolution_clock::now();
    hipMemcpy(c_gpu, dev_c, sizeof(float), hipMemcpyDeviceToHost);
    stop = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start);
    elapsed_time = duration.count() * 1.e-9;
    std::cout << "GPU time:" << elapsed_time << std::endl;

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    start = std::chrono::high_resolution_clock::now();
    dotCPU = dotproductCPU(a, b, size);
    stop = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start);
    elapsed_time2 = duration.count() * 1.e-9;
    std::cout << "CPU time:" << elapsed_time2 << std::endl;
    std::cout << "GPU speedup is x" << elapsed_time2 / elapsed_time << std::endl;
    std::cout << "GRIDSIZE:" << gridSize << std::endl;

    std::cout << "GPU results:" << c_gpu[0] << " / CPU results: " << dotCPU << std::endl;

    bool resultsMatch = (dotCPU - c_gpu[0]) < 1.0;
 
    if (resultsMatch)
    {
        std::cout << "GPU results match CPU results : " << dotCPU << std::endl;
    }
    else
    {
        std::cout << "GPU :(" << c_gpu[0] << ") do NOT!!!!! match CPU results : " << dotCPU << std::endl;
    }

    delete[] a;
    delete[] b;
    delete[] c_gpu;

    return 0;
}
